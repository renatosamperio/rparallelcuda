#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

__global__ void matrix_sum_kernel(const float* A, const float* B, const float* C, float* result, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // printf("  Executing kernel on (%d, %d)", idx, idy);
    if (idx < N && idy < N) {
        int index = idy * N + idx;
        result[index] = A[index] + B[index] + C[index];
    }
}

extern "C"
void matrix_sum_cuda(
    const float* A, 
    const float* B, 
    const float* C, 
    float* result, int N, int idx) {

    float *d_A, *d_B, *d_C, *d_result;
    size_t size = N * sizeof(float);

    CUDA_CHECK(hipMalloc(&d_A, size));
    CUDA_CHECK(hipMalloc(&d_B, size));
    CUDA_CHECK(hipMalloc(&d_C, size));
    CUDA_CHECK(hipMalloc(&d_result, size));

    CUDA_CHECK(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C, C, size, hipMemcpyHostToDevice));

    // dim3 threadsPerBlock(65536, 65536);
    // dim3 threadsPerBlock(1024, 1024);
    // dim3 threadsPerBlock(16, 16);
    dim3 threadsPerBlock(16, 4);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // std::cout << "    CUDA summing in: " << idx << std::endl;
    matrix_sum_kernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, d_result, N);
    CUDA_CHECK(hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipFree(d_result));
}
