#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <Rcpp.h>

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            Rcpp::Rcout << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            Rcpp::stop("Exiting..."); \
        } \
    } while (0)

__global__ void matrix_sum_kernel(const double* A, const double* B, const double* C, double* result, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // printf("  Executing kernel on (%d, %d)", idx, idy);
    if (idx < N && idy < N) {
        int index = idy * N + idx;
        result[index] = A[index] + B[index] + C[index];
    }
}

extern "C"
void matrix_sum_cuda(
    const double* A, 
    const double* B, 
    const double* C, 
    double* result, int N, int block1, int block2) {

    double *d_A, *d_B, *d_C, *d_result;
    size_t size = N * N * sizeof(double);

    CUDA_CHECK(hipMalloc(&d_A, size));
    CUDA_CHECK(hipMalloc(&d_B, size));
    CUDA_CHECK(hipMalloc(&d_C, size));
    CUDA_CHECK(hipMalloc(&d_result, size));

    CUDA_CHECK(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C, C, size, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(block1, block2);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_sum_kernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, d_result, N);
    CUDA_CHECK(hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipFree(d_result));
}


__global__ void matrixSumKernel(double* A, double* B, double* C, double* D, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < cols && idy < rows) {
        int index = idy * cols + idx;
        D[index] = A[index] + B[index] + C[index];
    }
}
